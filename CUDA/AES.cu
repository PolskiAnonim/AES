#include "hip/hip_runtime.h"
#include "AES.cuh"

extern int THREADS_PER_BLOCK;

void AES::RotWord(unsigned char* a) {
    unsigned char c = a[0];
    a[0] = a[1];
    a[1] = a[2];
    a[2] = a[3];
    a[3] = c;
}

void AES::SubWord(unsigned char* a) {
    for (int i = 0; i < 4; i++) {
        a[i] = sbox[a[i] / 16][a[i] % 16];
    }
}

void AES::Rcon(unsigned char* a, int n) {
    unsigned char c = 1;
    for (int i = 0; i < n - 1; i++) {
        c = (c << 1) ^ (((c >> 7) & 1) * 0x1b);
    }

    a[0] = c;
    a[1] = a[2] = a[3] = 0;
}

void AES::XorWords(unsigned char* a, unsigned char* b, unsigned char* c) {
    for (int i = 0; i < 4; i++) {
        c[i] = a[i] ^ b[i];
    }
}

void AES::KeyExpansion(const unsigned char key[], unsigned char w[]) {
    unsigned char temp[4];
    unsigned char rcon[4];

    int i = 0;
    while (i < 4 * 8) {
        w[i] = key[i];
        i++;
    }

    while (i < 16 * (14 + 1)) {
        temp[0] = w[i - 4 + 0];
        temp[1] = w[i - 4 + 1];
        temp[2] = w[i - 4 + 2];
        temp[3] = w[i - 4 + 3];

        if (i / 4 % 8 == 0) {
            RotWord(temp);
            SubWord(temp);
            Rcon(rcon, i / (8 * 4));
            XorWords(temp, rcon, temp);
        }
        else if (i / 4 % 8 == 4) {
            SubWord(temp);
        }

        w[i + 0] = w[i - 4 * 8] ^ temp[0];
        w[i + 1] = w[i + 1 - 4 * 8] ^ temp[1];
        w[i + 2] = w[i + 2 - 4 * 8] ^ temp[2];
        w[i + 3] = w[i + 3 - 4 * 8] ^ temp[3];
        i += 4;
    }
}

__device__ void SubBytes(unsigned char state[4][4]) {
    int i, j;
    unsigned char t;
    for (i = 0; i < 4; i++) {
        for (j = 0; j < 4; j++) {
            t = state[i][j];
            state[i][j] = sbox[t / 16][t % 16];
        }
    }
}

__device__ void ShiftRow(unsigned char state[4][4], int i, int n) {
    unsigned char tmp[4];
    int j = 0;
    for (; j < 4; j++) {
        tmp[j] = state[i][(j + n) % 4];
    }
    for (j = 0; j < 4; j++) {
        state[i][j] = tmp[j];
    }
}

__device__ void ShiftRows(unsigned char state[4][4]) {
    ShiftRow(state, 1, 1);
    ShiftRow(state, 2, 2);
    ShiftRow(state, 3, 3);
}

__device__ void MixColumns(unsigned char state[4][4]) {
    unsigned char temp_state[4][4] = {0};
    int i, j, k;
    for (i = 0; i < 4; ++i) {
        for (j = 0; j < 4; ++j) {
            for (k = 0; k < 4; ++k) {
                if (CMDS[i][j] == 1)
                    temp_state[i][k] ^= state[j][k];
                else
                    temp_state[i][k] ^= GF_MUL_TABLE[CMDS[i][j]][state[j][k]];
            }
        }
    }

    for (i = 0; i < 4; ++i) {
        for (j = 0; j < 4; ++j) {
            state[i][j] = temp_state[i][j];
        }
    }
}

__device__ void AddRoundKey(unsigned char state[4][4], unsigned char* key) {
    int i, j;
    for (i = 0; i < 4; i++) {
        for (j = 0; j < 4; j++) {
            state[i][j] ^= key[i + 4 * j];
        }
    }
}

__device__ void SubWord(unsigned char* a) {
    for (int i = 0; i < 4; i++) {
        a[i] = sbox[a[i] / 16][a[i] % 16];
    }
}

__device__ void RotWord(unsigned char* a) {
    unsigned char c = a[0];
    a[0] = a[1];
    a[1] = a[2];
    a[2] = a[3];
    a[3] = c;
}

__device__ void XorWords(unsigned char* a, unsigned char* b, unsigned char* c) {
    for (int i = 0; i < 4; i++) {
        c[i] = a[i] ^ b[i];
    }
}

__device__ void InvSubBytes(unsigned char state[4][4]) {
    int i, j;
    unsigned char t;
    for (i = 0; i < 4; i++) {
        for (j = 0; j < 4; j++) {
            t = state[i][j];
            state[i][j] = inv_sbox[t / 16][t % 16];
        }
    }
}

__device__ void InvMixColumns(unsigned char state[4][4]) {
    unsigned char temp_state[4][4] = { 0 };
    int i, j,k;
    for (i = 0; i < 4; ++i) {
        for (j = 0; j < 4; ++j) {
            for (k = 0; k < 4; ++k) {
                temp_state[i][k] ^= GF_MUL_TABLE[INV_CMDS[i][j]][state[j][k]];
            }
        }
    }

    for (i = 0; i < 4; ++i) {
        for (j = 0; j < 4; ++j) {
            state[i][j] = temp_state[i][j];
        }
    }
}

__device__ void InvShiftRows(unsigned char state[4][4]) {
    ShiftRow(state, 1, 4 - 1);
    ShiftRow(state, 2, 4 - 2);
    ShiftRow(state, 3, 4 - 3);
}

__device__ void XorBlocks(const unsigned char* a, const unsigned char* b,
    unsigned char* c, uint8_t len) {
    for (unsigned int i = 0; i < len; i++) {
        c[i] = a[i] ^ b[i];
    }
}

__global__ void EncryptBlock(const unsigned char in[], unsigned char out[],
    unsigned char* roundKeys, unsigned int len) {

    unsigned int offset = (blockIdx.x * blockDim.x + threadIdx.x)*16;
    if (offset >= len) return;
    unsigned char state[4][4];
    int i,j,round;

    for (i = 0; i < 4; i++) {
        for (j = 0; j < 4; j++) {
            state[i][j] = in[offset+i + 4 * j];
        }
    }

    AddRoundKey(state, roundKeys);

    for (round = 1; round <= 14-1; round++) {
        SubBytes(state);
        ShiftRows(state);
        MixColumns(state);
        AddRoundKey(state, roundKeys + round * 4 * 4);
    }

    SubBytes(state);
    ShiftRows(state);
    AddRoundKey(state, roundKeys + 14 * 4 * 4);

    for (i = 0; i < 4; i++) {
        for (j = 0; j < 4; j++) {
            out[offset+i + 4 * j] = state[i][j];
        }
    }
}

__global__ void DecryptBlock(const unsigned char in[], unsigned char out[],
    unsigned char* roundKeys, unsigned int len) {
    
    unsigned int offset = (blockIdx.x * blockDim.x + threadIdx.x) * 16;
    if (offset >= len) return;
    unsigned char state[4][4];
    int i, j, round;

    for (i = 0; i < 4; i++) {
        for (j = 0; j < 4; j++) {
            state[i][j] = in[offset + i + 4 * j];
        }
    }

    AddRoundKey(state, roundKeys + 14 * 4 * 4);

    for (round = 14 - 1; round >= 1; round--) {
        InvSubBytes(state);
        InvShiftRows(state);
        AddRoundKey(state, roundKeys + round * 4 * 4);
        InvMixColumns(state);
    }

    InvSubBytes(state);
    InvShiftRows(state);
    AddRoundKey(state, roundKeys);

    for (i = 0; i < 4; i++) {
        for (j = 0; j < 4; j++) {
            out[offset+i + 4 * j] = state[i][j];
        }
    }
}

unsigned char* AES::Encrypt(const unsigned char in[], unsigned int len, const unsigned char key[]) {
    unsigned char* out = new unsigned char[len];
    unsigned char* roundKeys = new unsigned char[4 * 4 * (14 + 1)];
    KeyExpansion(key, roundKeys);

    unsigned int numBlocks = (len + (THREADS_PER_BLOCK * 16 - 1)) / (THREADS_PER_BLOCK * 16);
    unsigned char* d_in;
    unsigned char* d_out;
    unsigned char* d_roundKeys;

    hipMalloc(&d_in, len);
    hipMalloc(&d_out, len);
    hipMalloc(&d_roundKeys, 4 * 4 * (14 + 1));

    hipMemcpyAsync(d_in, in, len, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_roundKeys, roundKeys, 4 * 4 * (14 + 1), hipMemcpyHostToDevice);

    EncryptBlock<<<numBlocks, THREADS_PER_BLOCK >>> (d_in, d_out, d_roundKeys,len);

    hipMemcpyAsync(out, d_out, len, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_roundKeys);
    delete[] roundKeys;

    return out;
}

unsigned char* AES::Decrypt(const unsigned char in[], unsigned int len, const unsigned char key[]) {
    unsigned char* out = new unsigned char[len];
    unsigned char* roundKeys = new unsigned char[4 * 4 * (14 + 1)];
    AES::KeyExpansion(key, roundKeys);

    unsigned int numBlocks = (len + (THREADS_PER_BLOCK * 16 - 1)) / (THREADS_PER_BLOCK * 16);
    unsigned char* d_in;
    unsigned char* d_out;
    unsigned char* d_roundKeys;

    hipMalloc(&d_in, len);
    hipMalloc(&d_out, len);
    hipMalloc(&d_roundKeys, 4 * 4 * (14 + 1));

    hipMemcpyAsync(d_in, in, len, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_roundKeys, roundKeys, 4 * 4 * (14 + 1), hipMemcpyHostToDevice);

    DecryptBlock <<<numBlocks, THREADS_PER_BLOCK >>> (d_in, d_out, d_roundKeys, len);

    hipMemcpyAsync(out, d_out, len, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_roundKeys);
    delete[] roundKeys;

    return out;
}

std::vector<unsigned char> AES::Encrypt(std::vector<unsigned char> in, std::vector<unsigned char> key) {
    unsigned char* out = Encrypt(in.data(), (unsigned int)in.size(), key.data());
    std::vector<unsigned char> v(out, out + in.size());
    delete[] out;
    return v;
}

std::vector<unsigned char> AES::Decrypt(std::vector<unsigned char> in, std::vector<unsigned char> key) {
    unsigned char* out = Decrypt(in.data(), (unsigned int)in.size(), key.data());
    std::vector<unsigned char> v(out, out + in.size());
    delete[] out;
    return v;
}