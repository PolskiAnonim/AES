﻿#include <iostream>
#include <chrono>
#include <string>
#include <fstream>

#include "Helpers.cuh"
#include "AES.cuh"

namespace main_functions {
    void encryption(std::vector<unsigned char>& key, std::string input_file_name, std::string output_option)
    {
        std::cout << "Reading file..." << std::endl;
        std::vector<unsigned char> data = read_file(input_file_name);
        std::cout << "Size: " << data.size() << " B" << std::endl;

        std::cout << "Encrypting data..." << std::endl;

        auto start_time = std::chrono::high_resolution_clock::now();
        std::vector<unsigned char> cipher = AES::Encrypt(data, key);
        auto end_time = std::chrono::high_resolution_clock::now();
        float enc_time = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1000.f;
        printf("Encryption time %.3f ms\n", enc_time);

        if (output_option != "") {
            std::cout << "Writing file..." << std::endl;
            write_hex_file(cipher, output_option);
        }
    }

    void decryption(std::vector<unsigned char>& key, std::string input_file_name, std::string output_option)
    {
        std::cout << "Reading file..." << std::endl;
        std::vector<unsigned char> data = read_hex_file(input_file_name);
        std::cout << "Size: " << data.size() << " B" << std::endl;

        std::cout << "Decrypting data..." << std::endl;

        auto start_time = std::chrono::high_resolution_clock::now();
        std::vector<unsigned char> plain = AES::Decrypt(data, key);
        auto end_time = std::chrono::high_resolution_clock::now();
        float dec_time = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1000.f;
        printf("Decryption time %.3f ms\n", dec_time);

        if (output_option != "") {
            std::cout << "Writing file..." << std::endl;
            write_file(plain, output_option);
        }
    }

    void both(std::vector<unsigned char>& key, std::string input_file_name) {
        std::cout << "Reading file..." << std::endl;
        std::vector<unsigned char> data = read_file(input_file_name);
        std::cout << "Size: " << data.size() << " B" << std::endl;

        std::cout << "Encrypting data..." << std::endl;

        auto start_time = std::chrono::high_resolution_clock::now();
        std::vector<unsigned char> cipher = AES::Encrypt(data, key);
        auto end_time = std::chrono::high_resolution_clock::now();
        float enc_time = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1000.f;
        printf("Encryption time %.3f ms\n", enc_time);

        std::cout << "Decrypting data..." << std::endl;
        start_time = std::chrono::high_resolution_clock::now();
        std::vector<unsigned char> plain = AES::Decrypt(cipher, key);
        end_time = std::chrono::high_resolution_clock::now();
        float dec_time = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1000.f;
        printf("Decryption time %.3f ms\n", dec_time);
        check_byte_arrays(data, plain);
    }
}

int main(int argc, char** argv) {
    // Ustawienie flag urządzenia - w zasadzie bardziej dla znormalizowania czasu pierwszego dostępu
    hipError_t cudaStatus = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDeviceFlags failed!");
        return 1;
    }


    if (argc < 4) {
        std::cerr << "Usage: " << argv[0] << "<operation> <key file> <input file> [<output option>]" << std::endl;
        std::cerr << "Where: <operation> - encrypt/decrypt/both, <output operation> - name of file to save or nothing" << std::endl;
        std::cerr << "(only for encrypt and decrypt)" << std::endl;
        return 1;
    }
    std::string operation = argv[1];
    std::vector<unsigned char> key = read_file(argv[2]);
    std::string input_file_name = argv[3];
    std::string output_option = (argc > 4) ? argv[4] : "";

    if (operation == "encrypt") {
        main_functions::encryption(key, input_file_name, output_option);
    }
    else if (operation == "decrypt") {
        main_functions::decryption(key, input_file_name, output_option);
    }
    else if (operation == "both") {
        main_functions::both(key, input_file_name);
    }
    else {
        std::cerr << "Invalid operation. Use 'encrypt' or 'decrypt' or 'both'." << std::endl;
        return 1;
    }

    return 0;
}
